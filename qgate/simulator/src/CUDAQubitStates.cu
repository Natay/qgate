#include "DeviceTypes.h"
#include "DeviceParallel.h"
#include "CUDAQubitStates.h"

#include <string.h>
#include <algorithm>

using namespace qgate_cuda;
using qgate::Qone;
using qgate::Qtwo;


template<class real>
void DeviceQubitStates<real>::allocate(const qgate::IdList &qregIdList) {
    deallocate();

    nQregIds_ = (int)qregIdList.size();
    nStates_ = Qone << nQregIds_;
    size_t qregIdListSize = sizeof(int) * nQregIds_;
    throwOnError(hipMalloc(&d_qregIdList_, qregIdListSize));
    throwOnError(hipMemcpy(d_qregIdList_, qregIdList.data(), qregIdListSize, hipMemcpyDefault));
    throwOnError(hipMalloc(&d_qstates_, sizeof(DeviceComplex) * nStates_));
}

template<class real>
void DeviceQubitStates<real>::deallocate() {
    if (d_qregIdList_ != NULL)
        throwOnError(hipFree(d_qregIdList_));
    if (d_qstates_ != NULL)
        throwOnError(hipFree(d_qstates_));
    d_qregIdList_ = NULL;
    d_qstates_ = NULL;
}

template<class real>
void DeviceQubitStates<real>::reset() {
    throwOnError(hipMemset(d_qstates_, 0, sizeof(DeviceComplex) * nStates_));
    DeviceComplex cQone(1.);
    throwOnError(hipMemcpy(d_qstates_, &cQone, sizeof(DeviceComplex), hipMemcpyDefault));
}

template<class real>
qgate::QstateIdx DeviceQubitStates<real>::getNStates() const {
    return Qone << nQregIds_;
}


template<class real>
CUDAQubitStates<real>::CUDAQubitStates() {
    if (sizeof(real) == sizeof(float))
        prec_ = qgate::precFP32;
    else
        prec_ = qgate::precFP64;
}

template<class real>
CUDAQubitStates<real>::~CUDAQubitStates() {
    deallocate();
}

template<class real>
void CUDAQubitStates<real>::allocate(const qgate::IdList &qregIdList) {
    
    qregIdList_ = qregIdList;
    devQstates_.allocate(qregIdList);
    devQstates_.reset();
}
    
template<class real>
void CUDAQubitStates<real>::deallocate() {
    devQstates_.deallocate();
}

template<class real>
void CUDAQubitStates<real>::reset() {
    devQstates_.reset();
}

template<class real>
int CUDAQubitStates<real>::getLane(int qregId) const {
    typename qgate::IdList::const_iterator it =
            std::find(qregIdList_.begin(), qregIdList_.end(), qregId);
    assert(it != qregIdList_.end());
    return (int)std::distance(qregIdList_.begin(), it);
}

template class CUDAQubitStates<float>;
template class CUDAQubitStates<double>;
