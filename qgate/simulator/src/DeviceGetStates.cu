#include "hip/hip_runtime.h"
#include "DeviceGetStates.h"
#include "DeviceFunctors.cuh"
#include "DeviceParallel.h"
#include <queue>

using namespace qgate_cuda;
using qgate::Qone;
using qgate::QstateIdx;
using qgate::QstateSize;


namespace {

template<class V> struct DeviceType;
template<> struct DeviceType<float> { typedef float Type; };
template<> struct DeviceType<double> { typedef double Type; };
template<> struct DeviceType<qgate::ComplexType<float>> { typedef DeviceComplexType<float> Type; };
template<> struct DeviceType<qgate::ComplexType<double>> { typedef DeviceComplexType<double> Type; };

}



template<class real>
DeviceGetStates<real>::DeviceGetStates(const qgate::QubitStatesList &qStatesList,
                                       CUDADeviceList &activeDevices) {

    activeDevices_ = activeDevices;
    int nQstates = (int)qStatesList.size();
    
    /* initialize list of qreg id list */
    IdList *idLists = new IdList[nQstates];
    DevicePtr *qStatesPtr = new DevicePtr[nQstates];
    memset(idLists, 0, sizeof(IdList) * nQstates);
    /* pack qreg id list */
    for (int qStatesIdx = 0; qStatesIdx < (int)nQstates; ++qStatesIdx) {
        const CUDAQubitStates<real> &cuQstates =
                static_cast<const CUDAQubitStates<real>&>(*qStatesList[qStatesIdx]);
        /* qregIds, qStatesPtr */
        const qgate::IdList &qregIds = cuQstates.getQregIdList();
        idLists[qStatesIdx].size = (int)qregIds.size();
        memcpy(idLists[qStatesIdx].id, qregIds.data(), sizeof(int) * qregIds.size());
        /* qstates ptr */
        qStatesPtr[qStatesIdx] = cuQstates.getDevicePtr();
    }
    /* create contexts */
    int nDevices = (int)activeDevices_.size();
    contexts_.resize(nDevices);
    
    for (int idx = 0; idx < nDevices; ++idx) {
        GetStatesContext &ctx = contexts_[idx];
        ctx.device = activeDevices_[idx];
        ctx.device->makeCurrent();
        ctx.dev.nQstates = nQstates;
        SimpleMemoryStore dmemStore = ctx.device->tempDeviceMemory();
        ctx.dev.d_idLists = dmemStore.allocate<IdList>(ctx.dev.nQstates);
        throwOnError(hipMemcpyAsync(ctx.dev.d_idLists, idLists,
                                     sizeof(IdList) * nQstates, hipMemcpyDefault));
        ctx.dev.d_qStatesPtr = dmemStore.allocate<DevicePtr>(ctx.dev.nQstates);
        throwOnError(hipMemcpyAsync(ctx.dev.d_qStatesPtr, qStatesPtr,
                                     sizeof(DevicePtr) * nQstates, hipMemcpyDefault));
    }
    for (int idx = 0; idx < (int)activeDevices_.size(); ++idx)
        activeDevices[idx]->synchronize();

    delete [] idLists;
    delete [] qStatesPtr;
}

template<class real>
DeviceGetStates<real>::~DeviceGetStates() {
    contexts_.clear();
}


template<class real>
void DeviceGetStates<real>::run(void *array, qgate::QstateIdx arrayOffset, qgate::MathOp op,
                                qgate::QstateIdx begin, qgate::QstateIdx end) {
    
    switch (op) {
    case qgate::mathOpNull: {
        Complex *cmpArray = static_cast<Complex*>(array);
        run(&cmpArray[arrayOffset], null<real>(), begin, end);
        break;
    }
    case qgate::mathOpProb: {
        real *vArray = static_cast<real*>(array);
        run(&vArray[arrayOffset], abs2<real>(), begin, end);
        break;
    }
    default:
        abort_("Unknown math op.");
    }
}


template<class real> template<class R, class F>
void DeviceGetStates<real>::run(R *values, const F &op,
                                qgate::QstateIdx begin, qgate::QstateIdx end) {
    typedef typename DeviceType<R>::Type DeviceR;

    std::queue<GetStatesContext*> running;

    
    SimpleMemoryStore hMemStore = contexts_[0].device->tempHostMemory();
    stride_ = (int)hMemStore.capacity<DeviceR>();
    for (int idx = 0; idx < (int)activeDevices_.size(); ++idx) {
        GetStatesContext &ctx = contexts_[idx];
        ctx.dev.h_values = hMemStore.allocate<DeviceR>(stride_);
    }
    
    begin_ = begin;
    pos_ = begin;
    end_ = end;
    
    /* FIXME: pipeline */
    for (int idx = 0; idx < (int)activeDevices_.size(); ++idx) {
        if (!launch<R, F>(contexts_[idx], op))
            break;
        running.push(&contexts_[idx]);
    }
    
    while (!running.empty()) {
        GetStatesContext *ctx = running.front();
        running.pop();
        syncAndCopy(values, *ctx);
        if (launch<R, F>(*ctx, op))
            running.push(ctx);
    }
}

template<class real> template<class R, class F>
bool DeviceGetStates<real>::launch(GetStatesContext &ctx, const F &op) {

    if (pos_ == end_)
        return false;

    ctx.device->makeCurrent();
    ctx.dev.begin = pos_;
    ctx.dev.end = std::min(pos_ + stride_, end_);
    
    DeviceGetStatesContext devCtx = ctx.dev;
    QstateIdx offset = ctx.dev.begin;
    
    typedef typename DeviceType<R>::Type DeviceR;
    auto calcStatesFunc = [=]__device__(QstateIdx globalIdx) {                 
        DeviceR v = DeviceR(1.);
        for (int iQstates = 0; iQstates < devCtx.nQstates; ++iQstates) {
            /* getStateByGlobalIdx() */
            const IdList &d_qregIds = devCtx.d_idLists[iQstates];
            QstateIdx localIdx = 0;
            for (int lane = 0; lane < d_qregIds.size; ++lane) {
                int qregId = d_qregIds.id[lane]; 
                if ((Qone << qregId) & globalIdx)
                    localIdx |= Qone << lane;
            }
            const DeviceComplex &state = devCtx.d_qStatesPtr[iQstates][localIdx];
            v *= op(state);
        }
        ((DeviceR*)devCtx.h_values)[globalIdx - offset] = v;
    };
    transform(ctx.dev.begin, ctx.dev.end, calcStatesFunc);
    pos_ = ctx.dev.end;

    return true;
}

template<class real> template<class R>
void DeviceGetStates<real>::syncAndCopy(R *values, GetStatesContext &ctx) {
    ctx.device->synchronize(); /* internally select this device. */
    memcpy(&values[ctx.dev.begin - begin_], ctx.dev.h_values, sizeof(R) * (ctx.dev.end - ctx.dev.begin));
}

template class DeviceGetStates<float>;
template class DeviceGetStates<double>;

