#include "hip/hip_runtime.h"
#include "DeviceGetStates.h"
#include "DeviceFunctors.cuh"
#include "DeviceParallel.h"
#include <queue>

using namespace qgate_cuda;
using qgate::Qone;
using qgate::QstateIdx;
using qgate::QstateSize;


namespace {

template<class V> struct DeviceType;
template<> struct DeviceType<float> { typedef float Type; };
template<> struct DeviceType<double> { typedef double Type; };
template<> struct DeviceType<qgate::ComplexType<float>> { typedef DeviceComplexType<float> Type; };
template<> struct DeviceType<qgate::ComplexType<double>> { typedef DeviceComplexType<double> Type; };

}



template<class real>
DeviceGetStates<real>::DeviceGetStates(const qgate::QubitStatesList &qStatesList,
                                       CUDADeviceList &activeDevices) {

    activeDevices_ = activeDevices;
    int nQstates = (int)qStatesList.size();
    
    /* initialize list of qreg id list */
    IdList *idLists = new IdList[nQstates];
    DevicePtr *qStatesPtr = new DevicePtr[nQstates];
    memset(idLists, 0, sizeof(IdList) * nQstates);
    /* pack qreg id list */
    for (int qStatesIdx = 0; qStatesIdx < (int)nQstates; ++qStatesIdx) {
        const CUDAQubitStates<real> &cuQstates =
                static_cast<const CUDAQubitStates<real>&>(*qStatesList[qStatesIdx]);
        /* qregIds, qStatesPtr */
        const qgate::IdList &qregIds = cuQstates.getQregIdList();
        idLists[qStatesIdx].size = (int)qregIds.size();
        memcpy(idLists[qStatesIdx].id, qregIds.data(), sizeof(int) * qregIds.size());
        /* qstates ptr */
        qStatesPtr[qStatesIdx] = cuQstates.getDevicePtr();
    }
    /* create contexts */
    contexts_ = new GetStatesContext[nQstates];
    memset(contexts_, 0, sizeof(GetStatesContext) * nQstates);
    
    for (int idx = 0; idx < (int)activeDevices_.size(); ++idx) {
        GetStatesContext &ctx = contexts_[idx];
        ctx.device = activeDevices_[idx];
        ctx.device->makeCurrent();
        ctx.nQstates = nQstates;
        SimpleMemoryStore dmemStore = ctx.device->tempDeviceMemory();
        ctx.d_idLists = dmemStore.allocate<IdList>(ctx.nQstates);
        throwOnError(hipMemcpyAsync(ctx.d_idLists, idLists,
                                     sizeof(IdList) * nQstates, hipMemcpyDefault));
        ctx.d_qStatesPtr = dmemStore.allocate<DevicePtr>(ctx.nQstates);
        throwOnError(hipMemcpyAsync(ctx.d_qStatesPtr, qStatesPtr,
                                     sizeof(DevicePtr) * nQstates, hipMemcpyDefault));
    }
    for (int idx = 0; idx < (int)activeDevices_.size(); ++idx)
        contexts_[idx].device->synchronize();

    delete [] idLists;
    delete [] qStatesPtr;
}

template<class real>
DeviceGetStates<real>::~DeviceGetStates() {
    delete [] contexts_;
}


template<class real>
void DeviceGetStates<real>::run(void *array, qgate::QstateIdx arrayOffset, qgate::MathOp op,
                                qgate::QstateIdx begin, qgate::QstateIdx end) {
    
    switch (op) {
    case qgate::mathOpNull: {
        Complex *cmpArray = static_cast<Complex*>(array);
        run(&cmpArray[arrayOffset], null<real>(), begin, end);
        break;
    }
    case qgate::mathOpProb: {
        real *vArray = static_cast<real*>(array);
        run(&vArray[arrayOffset], abs2<real>(), begin, end);
        break;
    }
    default:
        abort_("Unknown math op.");
    }
}


template<class real> template<class R, class F>
void DeviceGetStates<real>::run(R *values, const F &op,
                                qgate::QstateIdx begin, qgate::QstateIdx end) {
    typedef typename DeviceType<R>::Type DeviceR;

    std::queue<GetStatesContext*> running;

    
    SimpleMemoryStore hMemStore = contexts_[0].device->tempHostMemory();
    stride_ = (int)hMemStore.capacity<DeviceR>();
    for (int idx = 0; idx < (int)activeDevices_.size(); ++idx) {
        GetStatesContext &ctx = contexts_[idx];
        ctx.device->makeCurrent();
        ctx.h_values = hMemStore.allocate<DeviceR>(stride_);
    }
    
    begin_ = begin;
    pos_ = begin;
    end_ = end;
    
    /* FIXME: pipeline */
    for (int idx = 0; idx < (int)activeDevices_.size(); ++idx) {
        if (!launch<R, F>(contexts_[idx], op))
            break;
        running.push(&contexts_[idx]);
    }
    
    while (!running.empty()) {
        GetStatesContext *ctx = running.front();
        running.pop();
        syncAndCopy(values, *ctx);
        if (launch<R, F>(*ctx, op))
            running.push(ctx);
    }
}

template<class real> template<class R, class F>
bool DeviceGetStates<real>::launch(GetStatesContext &ctx, const F &op) {

    if (pos_ == end_)
        return false;

    ctx.begin = pos_;
    ctx.end = std::min(pos_ + stride_, end_);
    
    GetStatesContext devCtx = ctx;
    QstateIdx offset = ctx.begin;
    
    typedef typename DeviceType<R>::Type DeviceR;
    auto calcStatesFunc = [=]__device__(QstateIdx globalIdx) {                 
        DeviceR v = DeviceR(1.);
        for (int iQstates = 0; iQstates < devCtx.nQstates; ++iQstates) {
            /* getStateByGlobalIdx() */
            const IdList &d_qregIds = devCtx.d_idLists[iQstates];
            QstateIdx localIdx = 0;
            for (int lane = 0; lane < d_qregIds.size; ++lane) {
                int qregId = d_qregIds.id[lane]; 
                if ((Qone << qregId) & globalIdx)
                    localIdx |= Qone << lane;
            }
            const DeviceComplex &state = devCtx.d_qStatesPtr[iQstates][localIdx];
            v *= op(state);
        }
        ((DeviceR*)devCtx.h_values)[globalIdx - offset] = v;
    };
    transform(ctx.begin, ctx.end, calcStatesFunc);
    pos_ = ctx.end;

    return true;
}

template<class real> template<class R>
void DeviceGetStates<real>::syncAndCopy(R *values, GetStatesContext &ctx) {
    ctx.device->synchronize();
    memcpy(&values[ctx.begin - begin_], ctx.h_values, sizeof(R) * (ctx.end - ctx.begin));
}

template class DeviceGetStates<float>;
template class DeviceGetStates<double>;

