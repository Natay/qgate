#include "hip/hip_runtime.h"
#include "CUDAQubitProcessor.h"
#include "DeviceProcPrimitives.h"
#include "DeviceTypes.h"
#include "parallel.h"
#include "DeviceParallel.h"
#include <algorithm>

using namespace qgate_cuda;
using qgate::QstateIdx;
using qgate::QstateSize;


namespace {

template<class R>
struct abs2 {
    __device__ __forceinline__
    R operator()(const DeviceComplexType<R> &c) const {
        return c.real * c.real + c.imag * c.imag;
    }
};

template<class V>
struct null {
    __device__ __forceinline__
    const DeviceComplexType<V> &operator()(const DeviceComplexType<V> &c) const {
        return c;
    }
};

template<class V> struct DeviceType;
template<> struct DeviceType<float> { typedef float Type; };
template<> struct DeviceType<double> { typedef double Type; };
template<> struct DeviceType<qgate::ComplexType<float>> { typedef DeviceComplexType<float> Type; };
template<> struct DeviceType<qgate::ComplexType<double>> { typedef DeviceComplexType<double> Type; };
}


using qgate::Qone;
using qgate::Qtwo;


template<class real>
CUDAQubitProcessor<real>::CUDAQubitProcessor(CUDADevices &devices) : devices_(devices) { }

template<class real>
CUDAQubitProcessor<real>::~CUDAQubitProcessor() { }

template<class real>
void CUDAQubitProcessor<real>::initializeQubitStates(const qgate::IdList &qregIdList,
                                                     qgate::QubitStates &qstates) {
    CUQStates &cuQstates = static_cast<CUQStates&>(qstates);

    int nQregIds = (int)qregIdList.size();
    int nLanesInDevice = devices_.maxNLanesInDevice();
    int nRequiredDevices = 1 << (nQregIds - nLanesInDevice);
    if (devices_.size() < nRequiredDevices) {
        throwError("Number of GPUs is not enough, required = %d, current = %d.",
                   nRequiredDevices, devices_.size());
    }
    if (nRequiredDevices == 1)
        nLanesInDevice = (int)qregIdList.size();
    
    try {
        for (int idx = 0; idx < nRequiredDevices; ++idx) {
            CUDADevice &device = devices_[idx];
            deviceSet_.add(&device);
            procs_[idx] = new DeviceProcPrimitives<real>(device);
        }
        cuQstates.allocate(qregIdList, deviceSet_, nLanesInDevice);
    }
    catch (...) {
        finalizeQubitStates(qstates);
        throw;
    }
}

template<class real>
void CUDAQubitProcessor<real>::finalizeQubitStates(qgate::QubitStates &qstates) {
    CUQStates &cuQstates = static_cast<CUQStates&>(qstates);
    cuQstates.deallocate(deviceSet_);

    for (int idx = 0; idx < deviceSet_.size(); ++idx)
        delete procs_[idx];
}

template<class real>
void CUDAQubitProcessor<real>::resetQubitStates(qgate::QubitStates &qstates) {
    CUQStates &cuQstates = static_cast<CUQStates&>(qstates);
    typedef DeviceQubitStates<real> DeviceQstates;
    DeviceQstates &devQstates = cuQstates.getDeviceQubitStates();
    auto resetFunc = [=, &devQstates](int deviceIdx, QstateIdx begin, QstateIdx end) {
        procs_[deviceIdx]->fillZero(devQstates, begin, end);
    };
    apply(cuQstates, resetFunc);

    Complex cOne(1.);
    procs_[0]->set(devQstates, &cOne, 0, sizeof(cOne));
}

template<class real>
int CUDAQubitProcessor<real>::measure(double randNum,
                                      qgate::QubitStates &qstates, int qregId) {

    CUQStates &cuQstates = static_cast<CUQStates&>(qstates);
    typedef DeviceQubitStates<real> DeviceQstates;
    DeviceQstates &devQstates = cuQstates.getDeviceQubitStates();
    
    int cregValue = -1;

    int lane = cuQstates.getLane(qregId);

    auto traceOutLaunch = [=, &devQstates](int deviceIdx, QstateIdx begin, QstateIdx end) {
        procs_[deviceIdx]->traceOut_launch(devQstates, lane, begin, end);
    };
    apply(lane, cuQstates, traceOutLaunch);

    std::vector<real> partialSum(deviceSet_.size());
    auto traceOutSync = [=, &partialSum](int deviceIdx, QstateIdx begin, QstateIdx end) {
        partialSum[deviceIdx] = procs_[deviceIdx]->traceOut_sync();
    };
    apply(lane, cuQstates, traceOutSync);
    
    real prob = std::accumulate(partialSum.begin(), partialSum.end(), real(0.));

    /* reset bits */
    if (real(randNum) < prob) {
        cregValue = 0;
        auto set_0 = [=, &devQstates](int deviceIdx, QstateIdx begin, QstateIdx end){
            procs_[deviceIdx]->measure_set0(devQstates, lane, prob, begin, end);
        };
        applyLo(lane, cuQstates, set_0);
    }
    else {
        cregValue = 1;
        auto set_1 = [=, &devQstates](int deviceIdx, QstateIdx begin, QstateIdx end){
            procs_[deviceIdx]->measure_set1(devQstates, lane, prob, begin, end);
        };
        applyHi(lane, cuQstates, set_1);
    }
    deviceSet_.synchronize();

    return cregValue;
}


template<class real>
void CUDAQubitProcessor<real>::applyReset(qgate::QubitStates &qstates, int qregId) {
    
    CUQStates &cuQstates = static_cast<CUQStates&>(qstates);
    typedef DeviceQubitStates<real> DeviceQstates;
    DeviceQstates &devQstates = cuQstates.getDeviceQubitStates();
    
    int lane = cuQstates.getLane(qregId);
    
    auto reset = [=, &devQstates](int deviceIdx, QstateIdx begin, QstateIdx end) {
        procs_[deviceIdx]->applyReset(devQstates, lane, begin, end);
    };
    int nLanes = cuQstates.getNQregs();
    apply(lane, cuQstates, reset);
    deviceSet_.synchronize();
}


template<class real>
void CUDAQubitProcessor<real>::applyUnaryGate(const Matrix2x2C64 &mat, qgate::QubitStates &qstates, int qregId) {
    CUQStates &cuQstates = static_cast<CUQStates&>(qstates);
    typedef DeviceQubitStates<real> DeviceQstates;
    DeviceQstates &devQstates = cuQstates.getDeviceQubitStates();

    DeviceMatrix2x2C<real> dmat(mat);

    int lane = cuQstates.getLane(qregId);
    auto applyUnaryGate = [=, &devQstates](int deviceIdx, QstateIdx begin, QstateIdx end) {
        procs_[deviceIdx]->applyUnaryGate(dmat, devQstates, lane, begin, end);
    };
    int nLanes = cuQstates.getNQregs();
    apply(lane, cuQstates, applyUnaryGate);
    deviceSet_.synchronize(); /* must synchronize */
}


template<class real>
void CUDAQubitProcessor<real>::applyControlGate(const Matrix2x2C64 &mat, QubitStates &qstates,
                                                int controlId, int targetId) {
    CUQStates &cuQstates = static_cast<CUQStates&>(qstates);
    typedef DeviceQubitStates<real> DeviceQstates;
    DeviceQstates &devQstates = cuQstates.getDeviceQubitStates();

    DeviceMatrix2x2C<real> dmat(mat);
    int controlLane = cuQstates.getLane(controlId);
    int targetLane = cuQstates.getLane(targetId);

    auto applyControlGate = [=, &devQstates](int deviceIdx, QstateIdx begin, QstateIdx end) {
        procs_[deviceIdx]->applyControlGate(dmat, devQstates, controlLane, targetLane,
                                            begin, end);
    };
    int nLanes = cuQstates.getNQregs();
    applyHi(controlLane, cuQstates, applyControlGate);
}


template<class real> template<class F> void
CUDAQubitProcessor<real>::apply(CUQStates &cuQstates, const F &f) {
    int nLanes = cuQstates.getNQregs();
    QstateSize nThreads = Qone << nLanes;
    apply(cuQstates, f, 0, nThreads);
}

template<class real> template<class F> void
CUDAQubitProcessor<real>::apply(CUQStates &cuQstates, const F &f, QstateIdx begin, QstateIdx end) {
    int nLanes = cuQstates.getNQregs();
    QstateSize nThreads = Qone << nLanes;
    int nDevices = (int)deviceSet_.size();
    
    QstateSize nThreadsPerDevice = nThreads / nDevices;
    for (int iDevice = 0; iDevice < nDevices; ++iDevice) {
        QstateIdx beginInDevice = std::max(nThreadsPerDevice * iDevice, begin);
        QstateIdx endInDevice = std::min(nThreadsPerDevice * (iDevice + 1), end);
        if (beginInDevice != endInDevice)
            f(iDevice, beginInDevice, endInDevice);
    }
}

template<class real> template<class F> void CUDAQubitProcessor<real>::
apply(const qgate::IdList &lanes, CUQStates &cuQstates, F &f) {

    int nLanes = cuQstates.getNQregs();
    int nLanesInDevice = cuQstates.getNLanesInDevice();
    qgate::IdList bits;
    for (int idx = 0; idx < (int)lanes.size(); ++idx) {
        int lane = lanes[idx];
        if (nLanesInDevice <= lane) {
            int bit = 1 << (lane - nLanesInDevice);
            bits.push_back(bit);
        }
    }

    qgate::IdList ordered;
    
    int nDevicesPerGroup = 1 << bits.size();
    int nGroups = (int)deviceSet_.size() / nDevicesPerGroup;
    int nDevices = (int)deviceSet_.size();
    
    if (nDevicesPerGroup == 1) {
        for (int idx = 0; idx < nGroups; ++idx)
            ordered.push_back(idx);
    }
    else {
        for (int iGroup = 0; iGroup < nGroups; ++iGroup) {
            /* calculate base idx */
            int nBits = (int)bits.size();
            int mask = bits[0]- 1; /* mask_lo */
            int idx_base = iGroup | mask;
            for (int iBit = 0; iBit < nBits - 1; ++iBit) {
                int mask_lo = bits[iBit] - 1;
                int mask_hi = ~((bits[iBit + 1] << 1) - 1);
                idx_base |= (iGroup << iBit) & (mask_lo & mask_hi);
            }
            mask = ~((2 << bits.back()) - 1);
            idx_base |= (iGroup << (nBits - 1)) | mask;
            
            for (int idx = 0; idx < nDevicesPerGroup; ++idx) {
                int devIdx = idx_base;
                for (int iBit = 0; iBit < nBits; ++iBit) {
                    if (idx & iBit)
                        devIdx |= bits[iBit];
                }
                ordered.push_back(devIdx);
            }
        }
    }
    
    int nInputs = 1 << (int)lanes.size();
    QstateSize nThreads = (Qone << nLanes) / (1 << lanes.size());
    QstateSize nThreadsPerDevice = nThreads / nDevices;
    for (int iDevice = 0; iDevice < deviceSet_.size(); ++iDevice) {
        QstateIdx begin = nThreadsPerDevice * iDevice;
        QstateIdx end = nThreadsPerDevice * (iDevice + 1);
        f(ordered[iDevice], begin, end);
    }
}

template<class real> template<class F> void CUDAQubitProcessor<real>::
apply(int bitPos, CUQStates &cuQstates, const F &f) {
    int nLanes = cuQstates.getNQregs();
    QstateSize nThreads = (Qone << nLanes) / 2;
    apply(bitPos, cuQstates, f, nThreads, true, true);
}


template<class real> template<class F> void CUDAQubitProcessor<real>::
applyHi(int bitPos, CUQStates &cuQstates, const F &f) {
    int nLanes = cuQstates.getNQregs();
    QstateSize nThreads = (Qone << nLanes) / 4;
    apply(bitPos, cuQstates, f, nThreads, true, false);
}

template<class real> template<class F> void CUDAQubitProcessor<real>::
applyLo(int bitPos, CUQStates &cuQstates, const F &f) {
    int nLanes = cuQstates.getNQregs();
    QstateSize nThreads = (Qone << nLanes) / 4;
    apply(bitPos, cuQstates, f, nThreads, false, true);
}

template<class real> template<class F> void
CUDAQubitProcessor<real>::apply(int bitPos, CUQStates &cuQstates, const F &f,
                                qgate::QstateSize nThreads, bool runHi, bool runLo) {
    /* 1-bit gate */
    int nLanes = cuQstates.getNQregs();
    int nLanesInDevice = cuQstates.getNLanesInDevice();
    int nDevices = (int)deviceSet_.size();
    int nGroups;
    int bit;
    if (nLanesInDevice <= bitPos) {
        bit = 1 << (bitPos - nLanesInDevice);
        nGroups = nDevices / 2;
    }
    else {
        nGroups = nDevices;
        bit = 0;
    }

    qgate::IdList ordered;
    
    int nDevicesPerGroup = nDevices / nGroups;
    
    if (nDevicesPerGroup == 1) {
        for (int idx = 0; idx < nDevices; ++idx)
            ordered.push_back(idx);
    }
    else {
        for (int idx = 0; idx < nDevicesPerGroup; ++idx) {
            /* calculate base idx */
            int mask_0 = bit - 1;
            int mask_1 = ~((bit << 1) - 1);
            int idx_lo = ((idx << 1) | mask_1) & (idx & mask_0);
            if (runLo)
                ordered.push_back(idx_lo);
            int idx_hi = idx_lo | bit;
            if (runHi)
                ordered.push_back(idx_hi);
        }
    }
    
    QstateSize nThreadsPerDevice = nThreads / nDevices;
    for (int iDevice = 0; iDevice < nDevices; ++iDevice) {
        QstateIdx begin = nThreadsPerDevice * iDevice;
        QstateIdx end = nThreadsPerDevice * (iDevice + 1);
        f(ordered[iDevice], begin, end);
    }
}

/* get states */

template<class real>
void CUDAQubitProcessor<real>::getStates(void *array, QstateIdx arrayOffset,
                                         MathOp op,
                                         const QubitStatesList &qstatesList,
                                         QstateIdx beginIdx, QstateIdx endIdx) {

    for (int idx = 0; idx < (int)qstatesList.size(); ++idx) {
        const qgate::QubitStates *qstates = qstatesList[idx];
        if (sizeof(real) == sizeof(float))
            abortIf(qstates->getPrec() != qgate::precFP32, "Wrong type");
        else if (sizeof(real) == sizeof(double))
            abortIf(qstates->getPrec() != qgate::precFP64, "Wrong type");
    }

    
    typedef DeviceQubitStates<real> DeviceQstates;
    for (int idx = 0; idx < (int)qstatesList.size(); ++idx) {        
        CUQStates &cuQstates = static_cast<CUQStates&>(*qstatesList[idx]);
        DeviceQstates &devQstates = cuQstates.getDeviceQubitStates();
        auto getStatesFunc = [=, &devQstates](int deviceIdx, QstateIdx begin, QstateIdx end) {
            procs_[deviceIdx]->getStates(array, arrayOffset, op, devQstates, beginIdx, endIdx);
        };
        apply(cuQstates, getStatesFunc, beginIdx, endIdx);
    }
}


template class CUDAQubitProcessor<float>;
template class CUDAQubitProcessor<double>;
