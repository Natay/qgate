#include "hip/hip_runtime.h"
#include "CUDAQubitProcessor.h"
#include "DeviceProcPrimitives.h"
#include "DeviceTypes.h"
#include "parallel.h"
#include "DeviceParallel.h"
#include <algorithm>
#include <numeric>

using namespace qgate_cuda;
using qgate::QstateIdx;
using qgate::QstateSize;


namespace {

template<class R>
struct abs2 {
    __device__ __forceinline__
    R operator()(const DeviceComplexType<R> &c) const {
        return c.real * c.real + c.imag * c.imag;
    }
};

template<class V>
struct null {
    __device__ __forceinline__
    const DeviceComplexType<V> &operator()(const DeviceComplexType<V> &c) const {
        return c;
    }
};

template<class V> struct DeviceType;
template<> struct DeviceType<float> { typedef float Type; };
template<> struct DeviceType<double> { typedef double Type; };
template<> struct DeviceType<qgate::ComplexType<float>> { typedef DeviceComplexType<float> Type; };
template<> struct DeviceType<qgate::ComplexType<double>> { typedef DeviceComplexType<double> Type; };
}


using qgate::Qone;
using qgate::Qtwo;


template<class real>
CUDAQubitProcessor<real>::CUDAQubitProcessor(CUDADevices &devices) : devices_(devices) { }

template<class real>
CUDAQubitProcessor<real>::~CUDAQubitProcessor() { }

template<class real>
void CUDAQubitProcessor<real>::clear() {
    for (typename ProcMap::iterator it = procMap_.begin(); it != procMap_.end(); ++it)
        delete it->second;
    procMap_.clear();
}

template<class real>
void CUDAQubitProcessor<real>::prepare() {
    for (typename ProcMap::iterator it = procMap_.begin(); it != procMap_.end(); ++it)
        procMap_[it->first] = new DeviceProcPrimitives<real>(devices_[it->first]);
}

template<class real>
void CUDAQubitProcessor<real>::synchronize() {
    for (typename ProcMap::iterator it = procMap_.begin(); it != procMap_.end(); ++it)
        it->second->synchronize();
}

template<class real>
void CUDAQubitProcessor<real>::initializeQubitStates(const qgate::IdList &qregIdList,
                                                     qgate::QubitStates &qstates) {
    CUQStates &cuQstates = static_cast<CUQStates&>(qstates);

    int nQregIds = (int)qregIdList.size();
    int nLanesInDevice = devices_.maxNLanesInDevice();

    int nRequiredDevices;
    if (nLanesInDevice < nQregIds)
        nRequiredDevices = 1 << (nQregIds - nLanesInDevice);
    else
        nRequiredDevices = 1;

    if (devices_.size() < nRequiredDevices) {
        throwError("Number of GPUs is not enough, required = %d, current = %d.",
                   nRequiredDevices, devices_.size());
    }
    if (nRequiredDevices == 1)
        nLanesInDevice = (int)qregIdList.size();
    
    try {
        CUDADeviceList memoryOwners;
        for (int idx = 0; idx < nRequiredDevices; ++idx) {
            CUDADevice &device = devices_[idx];
            memoryOwners.push_back(&device);
            procMap_[device.getDeviceNumber()] = NULL;
        }
        cuQstates.allocate(qregIdList, memoryOwners, nLanesInDevice);
    }
    catch (...) {
        qstates.deallocate();
        throw;
    }
}

template<class real>
void CUDAQubitProcessor<real>::resetQubitStates(qgate::QubitStates &qstates) {
    CUQStates &cuQstates = static_cast<CUQStates&>(qstates);
    typedef DeviceQubitStates<real> DeviceQstates;
    DeviceQstates &devQstates = cuQstates.getDeviceQubitStates();
    auto resetFunc = [&](int deviceIdx, QstateIdx begin, QstateIdx end) {
                         int devIdx = cuQstates.getDeviceNumber(deviceIdx);
                         procMap_[devIdx]->fillZero(devQstates, begin, end);
                     };
    dispatchToDevices(cuQstates, resetFunc);

    Complex cOne(1.);
    procMap_[cuQstates.getDeviceNumber(0)]->set(devQstates, &cOne, 0, sizeof(cOne));
}

template<class real>
int CUDAQubitProcessor<real>::measure(double randNum,
                                      qgate::QubitStates &qstates, int qregId) {

    CUQStates &cuQstates = static_cast<CUQStates&>(qstates);
    typedef DeviceQubitStates<real> DeviceQstates;
    DeviceQstates &devQstates = cuQstates.getDeviceQubitStates();
    
    int cregValue = -1;

    int lane = cuQstates.getLane(qregId);

    auto traceOutLaunch = [&](int deviceIdx, QstateIdx begin, QstateIdx end) {
                              int devIdx = cuQstates.getDeviceNumber(deviceIdx);
                              procMap_[devIdx]->traceOut_launch(devQstates, lane, begin, end);
                          };
    apply(lane, cuQstates, traceOutLaunch);

    std::vector<real> partialSum(procMap_.size());
    auto traceOutSync = [&](int deviceIdx, QstateIdx begin, QstateIdx end) {
                            int devIdx = cuQstates.getDeviceNumber(deviceIdx);
                            partialSum[deviceIdx] = procMap_[devIdx]->traceOut_sync();
                        };
    apply(lane, cuQstates, traceOutSync);
    
    real prob = std::accumulate(partialSum.begin(), partialSum.end(), real(0.));

    /* reset bits */
    if (real(randNum) < prob) {
        cregValue = 0;
        auto set_0 = [&](int deviceIdx, QstateIdx begin, QstateIdx end){
                         int devIdx = cuQstates.getDeviceNumber(deviceIdx);
                         procMap_[devIdx]->measure_set0(devQstates, lane, prob, begin, end);
                     };
        apply(lane, cuQstates, set_0);
    }
    else {
        cregValue = 1;
        auto set_1 = [&](int deviceIdx, QstateIdx begin, QstateIdx end){
                         int devIdx = cuQstates.getDeviceNumber(deviceIdx);
                         procMap_[devIdx]->measure_set1(devQstates, lane, prob, begin, end);
                     };
        apply(lane, cuQstates, set_1);
    }
    synchronize();

    return cregValue;
}


template<class real>
void CUDAQubitProcessor<real>::applyReset(qgate::QubitStates &qstates, int qregId) {
    
    CUQStates &cuQstates = static_cast<CUQStates&>(qstates);
    typedef DeviceQubitStates<real> DeviceQstates;
    DeviceQstates &devQstates = cuQstates.getDeviceQubitStates();
    
    int lane = cuQstates.getLane(qregId);
    
    auto reset = [&](int deviceIdx, QstateIdx begin, QstateIdx end) {
                     int devIdx = cuQstates.getDeviceNumber(deviceIdx);
                     procMap_[devIdx]->applyReset(devQstates, lane, begin, end);
                 };
    int nLanes = cuQstates.getNQregs();
    apply(lane, cuQstates, reset);
    synchronize();
}


template<class real>
void CUDAQubitProcessor<real>::applyUnaryGate(const Matrix2x2C64 &mat, qgate::QubitStates &qstates, int qregId) {
    CUQStates &cuQstates = static_cast<CUQStates&>(qstates);
    typedef DeviceQubitStates<real> DeviceQstates;
    DeviceQstates &devQstates = cuQstates.getDeviceQubitStates();

    DeviceMatrix2x2C<real> dmat(mat);

    int lane = cuQstates.getLane(qregId);
    auto applyUnaryGate = [&](int deviceIdx, QstateIdx begin, QstateIdx end) {
                              int devIdx = cuQstates.getDeviceNumber(deviceIdx);
                              procMap_[devIdx]->applyUnaryGate(dmat, devQstates, lane, begin, end);
                          };
    int nLanes = cuQstates.getNQregs();
    apply(lane, cuQstates, applyUnaryGate);
    synchronize(); /* must synchronize */
}


template<class real>
void CUDAQubitProcessor<real>::applyControlGate(const Matrix2x2C64 &mat, QubitStates &qstates,
                                                int controlId, int targetId) {
    CUQStates &cuQstates = static_cast<CUQStates&>(qstates);
    typedef DeviceQubitStates<real> DeviceQstates;
    DeviceQstates &devQstates = cuQstates.getDeviceQubitStates();

    DeviceMatrix2x2C<real> dmat(mat);
    int controlLane = cuQstates.getLane(controlId);
    int targetLane = cuQstates.getLane(targetId);

    auto applyControlGate = [&](int deviceIdx, QstateIdx begin, QstateIdx end) {
                                int devIdx = cuQstates.getDeviceNumber(deviceIdx);
                                procMap_[devIdx]->applyControlGate(dmat, devQstates, controlLane, targetLane,
                                                                   begin, end);
                            };
    int nLanes = cuQstates.getNQregs();
    applyHi(controlLane, cuQstates, applyControlGate);

    synchronize();
}


template<class real> template<class F> void
CUDAQubitProcessor<real>::dispatchToDevices(CUQStates &cuQstates, const F &f) {
    int nLanes = cuQstates.getNQregs();
    QstateSize nThreads = Qone << nLanes;
    QstateSize nThreadsPerDevice = nThreads / procMap_.size();
    dispatchToDevices(cuQstates, f, 0, nThreads, nThreadsPerDevice);
}

template<class real> template<class F> void
CUDAQubitProcessor<real>::dispatchToDevices(CUQStates &cuQstates, const F &f, QstateIdx begin, QstateIdx end, QstateSize nThreadsPerDevice) {
    int nProcs = (int)procMap_.size();
    for (int iDevice = 0; iDevice < nProcs; ++iDevice) {
        QstateIdx beginInDevice = std::max(nThreadsPerDevice * iDevice, begin);
        QstateIdx endInDevice = std::min(nThreadsPerDevice * (iDevice + 1), end);
        if (beginInDevice != endInDevice)
            f(iDevice, beginInDevice, endInDevice);
    }
}

template<class real> template<class F> void CUDAQubitProcessor<real>::
apply(const qgate::IdList &lanes, CUQStates &cuQstates, F &f) {

    int nLanes = cuQstates.getNQregs();
    int nLanesInDevice = cuQstates.getNLanesInDevice();
    qgate::IdList bits;
    for (int idx = 0; idx < (int)lanes.size(); ++idx) {
        int lane = lanes[idx];
        if (nLanesInDevice <= lane) {
            int bit = 1 << (lane - nLanesInDevice);
            bits.push_back(bit);
        }
    }

    qgate::IdList ordered;
    
    int nProcsPerGroup = 1 << bits.size();
    int nGroups = (int)procMap_.size() / nProcsPerGroup;
    int nProcs = (int)procMap_.size();
    
    if (nProcsPerGroup == 1) {
        for (int idx = 0; idx < nGroups; ++idx)
            ordered.push_back(idx);
    }
    else {
        for (int iGroup = 0; iGroup < nGroups; ++iGroup) {
            /* calculate base idx */
            int nBits = (int)bits.size();
            int mask = bits[0]- 1; /* mask_lo */
            int idx_base = iGroup | mask;
            for (int iBit = 0; iBit < nBits - 1; ++iBit) {
                int mask_lo = bits[iBit] - 1;
                int mask_hi = ~((bits[iBit + 1] << 1) - 1);
                idx_base |= (iGroup << iBit) & (mask_lo & mask_hi);
            }
            mask = ~((2 << bits.back()) - 1);
            idx_base |= (iGroup << (nBits - 1)) | mask;
            
            for (int idx = 0; idx < nProcsPerGroup; ++idx) {
                int devIdx = idx_base;
                for (int iBit = 0; iBit < nBits; ++iBit) {
                    if (idx & iBit)
                        devIdx |= bits[iBit];
                }
                ordered.push_back(devIdx);
            }
        }
    }
    
    int nInputs = 1 << (int)lanes.size();
    QstateSize nThreads = (Qone << nLanes) / (1 << lanes.size());
    QstateSize nThreadsPerDevice = nThreads / nProcs;
    for (int iDevice = 0; iDevice < procMap_.size(); ++iDevice) {
        QstateIdx begin = nThreadsPerDevice * iDevice;
        QstateIdx end = nThreadsPerDevice * (iDevice + 1);
        f(ordered[iDevice], begin, end);
    }
}

template<class real> template<class F> void CUDAQubitProcessor<real>::
apply(int bitPos, CUQStates &cuQstates, const F &f) {
    int nLanes = cuQstates.getNQregs();
    QstateSize nThreads = (Qone << nLanes) / 2;
    apply(bitPos, cuQstates, f, nThreads, true, true);
}


template<class real> template<class F> void CUDAQubitProcessor<real>::
applyHi(int bitPos, CUQStates &cuQstates, const F &f) {
    int nLanes = cuQstates.getNQregs();
    QstateSize nThreads = (Qone << nLanes) / 4;
    apply(bitPos, cuQstates, f, nThreads, true, false);
}

template<class real> template<class F> void CUDAQubitProcessor<real>::
applyLo(int bitPos, CUQStates &cuQstates, const F &f) {
    int nLanes = cuQstates.getNQregs();
    QstateSize nThreads = (Qone << nLanes) / 4;
    apply(bitPos, cuQstates, f, nThreads, false, true);
}

template<class real> template<class F> void
CUDAQubitProcessor<real>::apply(int bitPos, CUQStates &cuQstates, const F &f,
                                qgate::QstateSize nThreads, bool runHi, bool runLo) {
    /* 1-bit gate */
    int nLanes = cuQstates.getNQregs();
    int nLanesInDevice = cuQstates.getNLanesInDevice();
    int nProcs = (int)procMap_.size();
    int nGroups;
    int bit;
    if (nLanesInDevice <= bitPos) {
        bit = 1 << (bitPos - nLanesInDevice);
        nGroups = nProcs / 2;
    }
    else {
        nGroups = nProcs;
        bit = 0;
    }

    qgate::IdList ordered;
    
    int nProcsPerGroup = nProcs / nGroups;
    
    if (nProcsPerGroup == 1) {
        for (int idx = 0; idx < nProcs; ++idx)
            ordered.push_back(idx);
    }
    else {
        for (int idx = 0; idx < nProcsPerGroup; ++idx) {
            /* calculate base idx */
            int mask_0 = bit - 1;
            int mask_1 = ~((bit << 1) - 1);
            int idx_lo = ((idx << 1) | mask_1) & (idx & mask_0);
            if (runLo)
                ordered.push_back(idx_lo);
            int idx_hi = idx_lo | bit;
            if (runHi)
                ordered.push_back(idx_hi);
        }
    }

    QstateSize nThreadsPerDevice = nThreads / nProcs;
    for (int iDevice = 0; iDevice < nProcs; ++iDevice) {
        QstateIdx begin = nThreadsPerDevice * iDevice;
        QstateIdx end = nThreadsPerDevice * (iDevice + 1);
        f(ordered[iDevice], begin, end);
    }
}

/* get states */

template<class real>
void CUDAQubitProcessor<real>::getStates(void *array, QstateIdx arrayOffset,
                                         MathOp op,
                                         const QubitStatesList &qstatesList,
                                         QstateIdx beginIdx, QstateIdx endIdx) {

    for (int idx = 0; idx < (int)qstatesList.size(); ++idx) {
        const qgate::QubitStates *qstates = qstatesList[idx];
        if (sizeof(real) == sizeof(float))
            abortIf(qstates->getPrec() != qgate::precFP32, "Wrong type");
        else if (sizeof(real) == sizeof(double))
            abortIf(qstates->getPrec() != qgate::precFP64, "Wrong type");
    }

    
    typedef DeviceQubitStates<real> DeviceQstates;
    for (int idx = 0; idx < (int)qstatesList.size(); ++idx) {        
        CUQStates &cuQstates = static_cast<CUQStates&>(*qstatesList[idx]);
        DeviceQstates &devQstates = cuQstates.getDeviceQubitStates();
        auto getStatesFunc = [&](int deviceIdx, QstateIdx begin, QstateIdx end) {
                                 int devNo = cuQstates.getDeviceNumber(deviceIdx);
                                 procMap_[devNo]->getStates(array, arrayOffset, op, devQstates, beginIdx, endIdx);
                             };
        QstateSize nThreadsPerDevice = (endIdx - beginIdx) / procMap_.size();
        dispatchToDevices(cuQstates, getStatesFunc, beginIdx, endIdx, nThreadsPerDevice);
    }
}


template class CUDAQubitProcessor<float>;
template class CUDAQubitProcessor<double>;
